#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <random>
#include <chrono>
#include <iomanip>

#define TILE_SIZE 32

__global__ void matrixMultiplyKernel(int* a, int* b, int* c, int m, int n, int k) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < m && col < k) {
		int sum = 0;
		for (int i = 0; i < n; i++) {
			sum += a[row * n + i] * b[i * k + col];
		}
		c[row * k + col] = sum;
	}
}

__global__ void matrixMultiplySharedKernel(int* a, int* b, int* c, int m, int n, int k) {
	
	__shared__ int sharedA[TILE_SIZE][TILE_SIZE];
	__shared__ int sharedB[TILE_SIZE][TILE_SIZE];

	
	int row = blockIdx.y * TILE_SIZE + threadIdx.y;
	int col = blockIdx.x * TILE_SIZE + threadIdx.x;

	int sum = 0;

	
	for (int phase = 0; phase < (n - 1) / TILE_SIZE + 1; ++phase) {
		
		if (row < m && phase * TILE_SIZE + threadIdx.x < n) {
			sharedA[threadIdx.y][threadIdx.x] = a[row * n + phase * TILE_SIZE + threadIdx.x];
		}
		else {
			sharedA[threadIdx.y][threadIdx.x] = 0;
		}

		if (col < k && phase * TILE_SIZE + threadIdx.y < n) {
			sharedB[threadIdx.y][threadIdx.x] = b[(phase * TILE_SIZE + threadIdx.y) * k + col];
		}
		else {
			sharedB[threadIdx.y][threadIdx.x] = 0;
		}

		__syncthreads();

		
		for (int i = 0; i < TILE_SIZE; ++i) {
			sum += sharedA[threadIdx.y][i] * sharedB[i][threadIdx.x];
		}

		__syncthreads();
	}

	
	if (row < m && col < k) {
		c[row * k + col] = sum;
	}
}


void allocateAndCopyToDevice(int** devicePtr, int* hostPtr, int size) {
	hipMalloc(devicePtr, size * sizeof(int));
	hipMemcpy(*devicePtr, hostPtr, size * sizeof(int), hipMemcpyHostToDevice);
}


void freeDeviceMemory(int* devicePtr) {
	hipFree(devicePtr);
}

void printMatrix(int* matrix, int rows, int cols) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			std::cout << std::setw(4) << matrix[i * cols + j];
		}
		std::cout << std::endl;
	}
}

int main() {
	const int m = 15000, n = 15000, k = 15000;
	int* a = new int[m * n];
	int* b = new int[n * k];
	int* c = new int[m * k];


	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> dis(0, 9);

	for (int i = 0; i < m * n; i++) {
		a[i] = dis(gen);
	}

	for (int i = 0; i < n * k; i++) {
		b[i] = dis(gen);
	}

	int* d_a, * d_b, * d_c;


	allocateAndCopyToDevice(&d_a, a, m * n);
	allocateAndCopyToDevice(&d_b, b, n * k);
	allocateAndCopyToDevice(&d_c, c, m * k);


	dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
	dim3 numBlocks((k + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);


	auto start = std::chrono::high_resolution_clock::now();
	
	//matrixMultiplyKernel << <numBlocks, threadsPerBlock >> > (d_a, d_b, d_c, m, n, k);
	matrixMultiplySharedKernel << <numBlocks, threadsPerBlock >> > (d_a, d_b, d_c, m, n, k);
	
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now();


	std::chrono::duration<double> elapsed = end - start;
	std::cout << "Time: " << elapsed.count() << " sec" << std::endl;


	//std::cout << "Matrix A:" << std::endl;
	//printMatrix(a, m, n);

	//std::cout << "Matrix B:" << std::endl;
	//printMatrix(b, n, k);


	//hipMemcpy(c, d_c, m * k * sizeof(int), hipMemcpyDeviceToHost);

	//std::cout << "Matrix C:" << std::endl;
	//printMatrix(c, m, k);


	freeDeviceMemory(d_a);
	freeDeviceMemory(d_b);
	freeDeviceMemory(d_c);
	delete[] a;
	delete[] b;
	delete[] c;


	return 0;
}